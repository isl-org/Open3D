// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#include "ATen/cuda/HIPContext.h"
#include "open3d/ml/impl/misc/ReduceSubarraysSum.cuh"
#include "open3d/ml/pytorch/TorchHelper.h"
#include "open3d/ml/pytorch/misc/ReduceSubarraysSumOpKernel.h"
#include "torch/script.h"

template <class T>
torch::Tensor ReduceSubarraysSumCUDA(const torch::Tensor& values,
                                     const torch::Tensor& row_splits) {
    auto device = values.device();
    torch::Tensor sums =
            torch::empty({row_splits.size(0) - 1},
                         torch::dtype(ToTorchDtype<T>()).device(device));

    auto stream = at::cuda::getCurrentCUDAStream();
    auto cuda_device_props = at::cuda::getCurrentDeviceProperties();
    open3d::ml::impl::ReduceSubarraysSumCUDA(
            stream, values.data_ptr<T>(), values.size(0),
            row_splits.data_ptr<int64_t>(), row_splits.size(0) - 1,
            sums.data_ptr<T>());
    return sums;
}
#define INSTANTIATE(T)                                                     \
    template torch::Tensor ReduceSubarraysSumCUDA<T>(const torch::Tensor&, \
                                                     const torch::Tensor&);

INSTANTIATE(int32_t)
INSTANTIATE(int64_t)
INSTANTIATE(float)
INSTANTIATE(double)
