// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#include "ATen/cuda/HIPContext.h"
#include "open3d/ml/impl/misc/InvertNeighborsList.cuh"
#include "open3d/ml/pytorch/TorchHelper.h"
#include "open3d/ml/pytorch/misc/InvertNeighborsListOpKernel.h"
#include "torch/script.h"

template <class TIndex, class TAttr>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> InvertNeighborsListCUDA(
        int64_t num_points,
        const torch::Tensor& inp_neighbors_index,
        const torch::Tensor& inp_neighbors_row_splits,
        const torch::Tensor& inp_neighbors_attributes) {
    auto device = inp_neighbors_index.device();
    torch::Tensor neighbors_index =
            torch::empty(inp_neighbors_index.sizes(),
                         torch::dtype(ToTorchDtype<TIndex>()).device(device));
    torch::Tensor neighbors_row_splits = torch::empty(
            {num_points + 1}, torch::dtype(torch::kInt64).device(device));
    torch::Tensor neighbors_attributes =
            torch::empty_like(inp_neighbors_attributes);

    auto stream = at::cuda::getCurrentCUDAStream();
    auto cuda_device_props = at::cuda::getCurrentDeviceProperties();
    const int texture_alignment = cuda_device_props->textureAlignment;

    int num_attributes;
    if (inp_neighbors_attributes.size(0) == 0) {
        num_attributes = 0;
    } else {
        num_attributes = 1;
        for (int i = 1; i < inp_neighbors_attributes.dim(); ++i)
            num_attributes *= inp_neighbors_attributes.size(i);
    }

    void* temp_ptr = nullptr;
    size_t temp_size = 0;

    // determine temp_size
    open3d::ml::impl::InvertNeighborsListCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            inp_neighbors_index.data_ptr<TIndex>(),
            num_attributes ? inp_neighbors_attributes.data_ptr<TAttr>()
                           : nullptr,
            num_attributes,
            (int64_t*)inp_neighbors_row_splits.data_ptr<int64_t>(),
            inp_neighbors_row_splits.size(0) - 1,
            neighbors_index.data_ptr<TIndex>(),
            num_attributes ? neighbors_attributes.data_ptr<TAttr>() : nullptr,
            neighbors_index.size(0),
            (int64_t*)neighbors_row_splits.data_ptr<int64_t>(),
            neighbors_row_splits.size(0) - 1);

    auto temp_tensor = CreateTempTensor(temp_size, device, &temp_ptr);

    // actually invert the list
    open3d::ml::impl::InvertNeighborsListCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            inp_neighbors_index.data_ptr<TIndex>(),
            num_attributes ? inp_neighbors_attributes.data_ptr<TAttr>()
                           : nullptr,
            num_attributes,
            (int64_t*)inp_neighbors_row_splits.data_ptr<int64_t>(),
            inp_neighbors_row_splits.size(0) - 1,
            neighbors_index.data_ptr<TIndex>(),
            num_attributes ? neighbors_attributes.data_ptr<TAttr>() : nullptr,
            neighbors_index.size(0),
            (int64_t*)neighbors_row_splits.data_ptr<int64_t>(),
            neighbors_row_splits.size(0) - 1);

    return std::make_tuple(neighbors_index, neighbors_row_splits,
                           neighbors_attributes);
}
#define INSTANTIATE(TIndex, TAttr)                                        \
    template std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>      \
    InvertNeighborsListCUDA<TIndex, TAttr>(int64_t, const torch::Tensor&, \
                                           const torch::Tensor&,          \
                                           const torch::Tensor&);

INSTANTIATE(int32_t, uint8_t)
INSTANTIATE(int32_t, int8_t)
INSTANTIATE(int32_t, int16_t)
INSTANTIATE(int32_t, int32_t)
INSTANTIATE(int32_t, int64_t)
INSTANTIATE(int32_t, float)
INSTANTIATE(int32_t, double)
