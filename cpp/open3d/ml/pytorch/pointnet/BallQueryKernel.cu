// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//***************************************************************************************/
//
//    Based on Pointnet2 Library (MIT License):
//    https://github.com/sshaoshuai/Pointnet2.PyTorch
//
//    Copyright (c) 2019 Shaoshuai Shi
//
//    Permission is hereby granted, free of charge, to any person obtaining a
//    copy of this software and associated documentation files (the "Software"),
//    to deal in the Software without restriction, including without limitation
//    the rights to use, copy, modify, merge, publish, distribute, sublicense,
//    and/or sell copies of the Software, and to permit persons to whom the
//    Software is furnished to do so, subject to the following conditions:
//
//    The above copyright notice and this permission notice shall be included in
//    all copies or substantial portions of the Software.
//
//    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
//    THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
//    FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
//    DEALINGS IN THE SOFTWARE.
//
//***************************************************************************************/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "ATen/cuda/HIPContext.h"
#include "open3d/ml/contrib/BallQuery.cuh"
#include "open3d/ml/contrib/cuda_utils.h"
#include "open3d/ml/pytorch/pointnet/BallQueryKernel.h"

using namespace open3d::ml::contrib;

void ball_query_launcher(int b,
                         int n,
                         int m,
                         float radius,
                         int nsample,
                         const float *new_xyz,
                         const float *xyz,
                         int *idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)

    hipError_t err;

    auto stream = at::cuda::getCurrentCUDAStream();

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK),
                b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    ball_query_kernel<<<blocks, threads, 0, stream>>>(b, n, m, radius, nsample,
                                                      new_xyz, xyz, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
