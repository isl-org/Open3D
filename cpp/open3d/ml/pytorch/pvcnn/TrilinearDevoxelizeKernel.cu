// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//
//    Based on PVCNN Library (MIT License):
//    https://github.com/mit-han-lab/pvcnn
//
// Copyright (c) 2018 Zhijian Liu, Haotian Tang, Yujun Lin
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ----------------------------------------------------------------------------

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "ATen/cuda/HIPContext.h"
#include "open3d/ml/contrib/TrilinearDevoxelize.cuh"
#include "open3d/ml/contrib/cuda_utils.h"
#include "open3d/ml/pytorch/pvcnn/TrilinearDevoxelizeKernel.h"

using namespace open3d::ml::contrib;

void TrilinearDevoxelize(int b,
                         int c,
                         int n,
                         int r,
                         int r2,
                         int r3,
                         bool training,
                         const float *coords,
                         const float *feat,
                         int *inds,
                         float *wgts,
                         float *outs) {
    hipError_t err;

    auto stream = at::cuda::getCurrentCUDAStream();

    TrilinearDevoxelizeKernel<<<b, OptNumThreads(n), 0, stream>>>(
            b, c, n, r, r2, r3, training, coords, feat, inds, wgts, outs);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

void TrilinearDevoxelizeGrad(int b,
                             int c,
                             int n,
                             int r3,
                             const int *inds,
                             const float *wgts,
                             const float *grad_y,
                             float *grad_x) {
    hipError_t err;

    auto stream = at::cuda::getCurrentCUDAStream();

    TrilinearDevoxelizeGradKernel<<<b, OptNumThreads(n), 0, stream>>>(
            b, c, n, r3, inds, wgts, grad_y, grad_x);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
