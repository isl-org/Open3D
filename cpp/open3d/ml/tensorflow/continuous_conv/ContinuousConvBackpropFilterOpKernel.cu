#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#define EIGEN_USE_GPU
#include "ContinuousConvBackpropFilterOpKernel.h"
#include "open3d/core/CUDAUtils.h"
#include "open3d/ml/impl/continuous_conv/ContinuousConvBackpropFilter.cuh"

using namespace open3d;
using namespace open3d::ml;
using namespace open3d::ml::impl;
using namespace tensorflow;

template <class TFeat, class TOut, class TReal, class TIndex>
class ContinuousConvBackpropFilterOpKernelCUDA
    : public ContinuousConvBackpropFilterOpKernel<TIndex> {
public:
    explicit ContinuousConvBackpropFilterOpKernelCUDA(
            OpKernelConstruction* construction)
        : ContinuousConvBackpropFilterOpKernel<TIndex>(construction) {
        texture_alignment =
                open3d::core::GetCUDACurrentDeviceTextureAlignment();
    }

    void Kernel(tensorflow::OpKernelContext* context,
                const tensorflow::Tensor& filter,
                const tensorflow::Tensor& out_positions,
                const tensorflow::Tensor& extents,
                const tensorflow::Tensor& offset,
                const tensorflow::Tensor& inp_positions,
                const tensorflow::Tensor& inp_features,
                const tensorflow::Tensor& inp_importance,
                const tensorflow::Tensor& neighbors_index,
                const tensorflow::Tensor& neighbors_importance,
                const tensorflow::Tensor& neighbors_row_splits,
                const tensorflow::Tensor& out_features_gradient,
                const std::vector<int>& filter_dims,
                const bool individual_extents,
                const bool isotropic_extents,
                const bool point_importances,
                const bool has_neighbors_importances,
                tensorflow::Tensor& filter_backprop) {
        auto device = context->eigen_gpu_device();

        void* temp_ptr = nullptr;
        size_t temp_size = 0;
        size_t max_temp_size = 0;

        // determine temp_size
        CConvBackpropFilterCUDA<TFeat, TOut, TReal, TIndex>(
                device.stream(), temp_ptr, temp_size, max_temp_size,
                texture_alignment, filter_backprop.flat<TOut>().data(),
                filter_dims, out_positions.shape().dim_size(0),
                out_positions.flat<TReal>().data(),
                inp_positions.shape().dim_size(0),
                inp_positions.flat<TReal>().data(),
                inp_features.flat<TFeat>().data(),
                point_importances ? inp_importance.flat<TFeat>().data()
                                  : nullptr,
                neighbors_index.shape().dim_size(0),
                (TIndex*)neighbors_index.flat<TIndex>().data(),
                has_neighbors_importances
                        ? neighbors_importance.flat<TFeat>().data()
                        : nullptr,
                (int64_t*)neighbors_row_splits.flat<int64>().data(),
                extents.flat<TReal>().data(), offset.flat<TReal>().data(),
                out_features_gradient.flat<TFeat>().data(), this->interpolation,
                this->coordinate_mapping, this->align_corners,
                individual_extents, isotropic_extents, this->normalize);

        temp_size =
                std::max(std::min(size_t(this->max_temp_mem_MB) * 1024 * 1024,
                                  max_temp_size),
                         temp_size);

        Tensor temp_tensor;
        TensorShape temp_shape({ssize_t(temp_size)});
        OP_REQUIRES_OK(context,
                       context->allocate_temp(DataTypeToEnum<uint8_t>::v(),
                                              temp_shape, &temp_tensor));
        temp_ptr = temp_tensor.flat<uint8_t>().data();

        // actually run the operation
        CConvBackpropFilterCUDA<TFeat, TOut, TReal, TIndex>(
                device.stream(), temp_ptr, temp_size, max_temp_size,
                texture_alignment, filter_backprop.flat<TOut>().data(),
                filter_dims, out_positions.shape().dim_size(0),
                out_positions.flat<TReal>().data(),
                inp_positions.shape().dim_size(0),
                inp_positions.flat<TReal>().data(),
                inp_features.flat<TFeat>().data(),
                point_importances ? inp_importance.flat<TFeat>().data()
                                  : nullptr,
                neighbors_index.shape().dim_size(0),
                (TIndex*)neighbors_index.flat<TIndex>().data(),
                has_neighbors_importances
                        ? neighbors_importance.flat<TFeat>().data()
                        : nullptr,
                (int64_t*)neighbors_row_splits.flat<int64>().data(),
                extents.flat<TReal>().data(), offset.flat<TReal>().data(),
                out_features_gradient.flat<TFeat>().data(), this->interpolation,
                this->coordinate_mapping, this->align_corners,
                individual_extents, isotropic_extents, this->normalize);
    }

private:
    int texture_alignment;
};

#define REG_KB(feattype, outtype, realtype, indextype)                  \
    REGISTER_KERNEL_BUILDER(                                            \
            Name("Open3DContinuousConvBackpropFilter")                  \
                    .Device(DEVICE_GPU)                                 \
                    .TypeConstraint<feattype>("TFeat")                  \
                    .TypeConstraint<outtype>("output_type")             \
                    .TypeConstraint<realtype>("TReal")                  \
                    .TypeConstraint<indextype>("TIndex"),               \
            ContinuousConvBackpropFilterOpKernelCUDA<feattype, outtype, \
                                                     realtype, indextype>);
REG_KB(float, float, float, int32)
#undef REG_KB
