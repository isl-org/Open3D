// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#define EIGEN_USE_GPU
#include "TrilinearDevoxelizeKernel.h"
#include "open3d/ml/Helper.h"
#include "open3d/ml/contrib/TrilinearDevoxelize.cuh"
#include "open3d/ml/contrib/cuda_utils.h"

using namespace open3d;
using namespace open3d::ml;
using namespace open3d::ml::contrib;
using namespace tensorflow;

class TrilinearDevoxelizeOpKernelCUDA : public TrilinearDevoxelizeOpKernel {
public:
    explicit TrilinearDevoxelizeOpKernelCUDA(OpKernelConstruction* context)
        : TrilinearDevoxelizeOpKernel(context) {}

    void Kernel(tensorflow::OpKernelContext* context,
                int b,
                int c,
                int n,
                int r,
                int r2,
                int r3,
                bool training,
                const float* coords,
                const float* feat,
                int* inds,
                float* wgts,
                float* outs) {
        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        TrilinearDevoxelizeKernel<<<b, OptNumThreads(n), 0, stream>>>(
                b, c, n, r, r2, r3, training, coords, feat, inds, wgts, outs);

        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("Open3DTrilinearDevoxelize").Device(DEVICE_GPU),
                        TrilinearDevoxelizeOpKernelCUDA)

class TrilinearDevoxelizeGradOpKernelCUDA
    : public TrilinearDevoxelizeGradOpKernel {
public:
    explicit TrilinearDevoxelizeGradOpKernelCUDA(OpKernelConstruction* context)
        : TrilinearDevoxelizeGradOpKernel(context) {}

    void Kernel(tensorflow::OpKernelContext* context,
                int b,
                int c,
                int n,
                int r3,
                const int* inds,
                const float* wgts,
                const float* grad_y,
                float* grad_x) {
        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        TrilinearDevoxelizeGradKernel<<<b, OptNumThreads(n), 0, stream>>>(
                b, c, n, r3, inds, wgts, grad_y, grad_x);

        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(
        Name("Open3DTrilinearDevoxelizeGrad").Device(DEVICE_GPU),
        TrilinearDevoxelizeGradOpKernelCUDA)
