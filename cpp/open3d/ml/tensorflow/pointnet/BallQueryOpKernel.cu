// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#define EIGEN_USE_GPU
#include "BallQueryOpKernel.h"
#include "open3d/ml/Helper.h"
#include "open3d/ml/contrib/BallQuery.cuh"
#include "open3d/ml/contrib/cuda_utils.h"

using namespace open3d;
using namespace open3d::ml;
using namespace open3d::ml::contrib;
using namespace tensorflow;

class BallQueryOpKernelCUDA : public BallQueryOpKernel {
public:
    explicit BallQueryOpKernelCUDA(OpKernelConstruction *construction)
        : BallQueryOpKernel(construction) {}

    void Kernel(tensorflow::OpKernelContext *context,
                int b,
                int n,
                int m,
                float radius,
                int nsample,
                const float *new_xyz,
                const float *xyz,
                int *idx) {
        // dataset: (B, N, 3)
        // tmp: (B, N)
        // output:
        //      idx: (B, M)

        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        dim3 blocks(DIVUP(m, THREADS_PER_BLOCK),
                    b);  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);

        ball_query_kernel<<<blocks, threads, 0, stream>>>(
                b, n, m, radius, nsample, new_xyz, xyz, idx);
        // hipDeviceSynchronize();  // for using printf in kernel function
        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("Open3DBallQuery").Device(DEVICE_GPU),
                        BallQueryOpKernelCUDA);
