#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#include <sstream>
#include <vector>

#include "open3d/ml/impl/sparse_conv/SparseConv.cuh"
#include "open3d/ml/paddle/PaddleHelper.h"

using namespace open3d::ml::impl;

template <class TFeat, class TOut, class TIndex, class TKernelIndex>
void SparseConvCUDA(const paddle::Tensor& filters,
                    const paddle::Tensor& inp_features,
                    const paddle::Tensor& inp_importance,
                    const paddle::Tensor& neighbors_index,
                    const paddle::Tensor& neighbors_kernel_index,
                    const paddle::Tensor& neighbors_importance,
                    const paddle::Tensor& neighbors_row_splits,
                    const bool normalize,
                    const int64_t max_temp_mem_MB,
                    paddle::Tensor& out_features) {
    std::vector<int> filter_dims;
    for (auto d : filters.shape()) {
        filter_dims.push_back(static_cast<int>(d));
    }

    auto stream = filters.stream();
    // -1 means current global place
    auto cuda_device_props = phi::backends::gpu::GetDeviceProperties(-1);
    const int texture_alignment = cuda_device_props.textureAlignment;

    auto place = filters.place();

    void* temp_ptr = nullptr;
    size_t temp_size = 0;
    size_t max_temp_size = 0;

    // determine temp_size
    SparseConvComputeFeaturesCUDA<TFeat, TOut, TIndex, TKernelIndex>(
            stream, temp_ptr, temp_size, max_temp_size, texture_alignment,
            out_features.data<TOut>(), filter_dims, filters.data<TFeat>(),
            neighbors_row_splits.shape()[0] - 1, inp_features.shape()[0],
            inp_features.data<TFeat>(),
            inp_importance.shape()[0] ? inp_importance.data<TFeat>() : nullptr,
            neighbors_index.shape()[0], neighbors_index.data<TIndex>(),
            neighbors_kernel_index.data<TKernelIndex>(),
            neighbors_importance.shape()[0] ? neighbors_importance.data<TFeat>()
                                            : nullptr,
            neighbors_row_splits.data<int64_t>(), normalize);

    temp_size = std::max(
            std::min(static_cast<size_t>(max_temp_mem_MB) * 1024 * 1024,
                     max_temp_size),
            temp_size);

    auto temp_tensor = CreateTempTensor(temp_size, place, &temp_ptr);

    // actually run the operation
    SparseConvComputeFeaturesCUDA<TFeat, TOut, TIndex, TKernelIndex>(
            stream, temp_ptr, temp_size, max_temp_size, texture_alignment,
            out_features.data<TOut>(), filter_dims, filters.data<TFeat>(),
            neighbors_row_splits.shape()[0] - 1, inp_features.shape()[0],
            inp_features.data<TFeat>(),
            inp_importance.shape()[0] ? inp_importance.data<TFeat>() : nullptr,
            neighbors_index.shape()[0], neighbors_index.data<TIndex>(),
            neighbors_kernel_index.data<TKernelIndex>(),
            neighbors_importance.shape()[0] ? neighbors_importance.data<TFeat>()
                                            : nullptr,
            neighbors_row_splits.data<int64_t>(), normalize);
}
#define INSTANTIATE(TFeat, TOut, TReal, TIndex)                                \
    template void SparseConvCUDA<TFeat, TOut, TReal, TIndex>(                  \
            const paddle::Tensor& filters, const paddle::Tensor& inp_features, \
            const paddle::Tensor& inp_importance,                              \
            const paddle::Tensor& neighbors_index,                             \
            const paddle::Tensor& neighbors_kernel_index,                      \
            const paddle::Tensor& neighbors_importance,                        \
            const paddle::Tensor& neighbors_row_splits, const bool normalize,  \
            const int64_t max_temp_mem_MB, paddle::Tensor& out_features);

INSTANTIATE(float, float, int32_t, uint8_t)
