#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "open3d/ml/contrib/IoU.h"
#include "open3d/ml/contrib/IoUImpl.h"

namespace open3d {
namespace ml {
namespace contrib {

static constexpr int block_size = 128;
static constexpr int thread_size = 4;

__global__ void IoUBevElementKernel(const float *boxes_a,
                                    const float *boxes_b,
                                    float *iou,
                                    int num_a,
                                    int num_b) {
    // Using the "grid-stride loops" pattern.
    int n = num_a * num_b;
    int items_per_block = block_size * thread_size;
    int idx = blockIdx.x * items_per_block + threadIdx.x;
#pragma unroll
    for (int i = 0; i < thread_size; i++) {
        if (idx < n) {
            int idx_a = idx / num_b;
            int idx_b = idx % num_b;

            const float *box_a = boxes_a + idx_a * 5;
            const float *box_b = boxes_b + idx_b * 5;
            float *out = iou + idx_a * num_b + idx_b;
            *out = IoUBev2DWithCenterAndSize(box_a, box_b);

            idx += block_size;
        }
    }
}

void IoUBevCUDAKernel(const float *boxes_a,
                      const float *boxes_b,
                      float *iou,
                      int num_a,
                      int num_b) {
    int n = num_a * num_b;
    if (n == 0) {
        return;
    }
    int items_per_block = block_size * thread_size;
    int grid_size = (n + items_per_block - 1) / items_per_block;
    IoUBevElementKernel<<<grid_size, block_size>>>(boxes_a, boxes_b, iou, num_a,
                                                   num_b);
}

__global__ void IoU3dElementKernel(const float *boxes_a,
                                   const float *boxes_b,
                                   float *iou,
                                   int num_a,
                                   int num_b) {
    // Using the "grid-stride loops" pattern.
    int n = num_a * num_b;
    int items_per_block = block_size * thread_size;
    int idx = blockIdx.x * items_per_block + threadIdx.x;
#pragma unroll
    for (int i = 0; i < thread_size; i++) {
        if (idx < n) {
            int idx_a = idx / num_b;
            int idx_b = idx % num_b;

            const float *box_a = boxes_a + idx_a * 7;
            const float *box_b = boxes_b + idx_b * 7;
            float *out = iou + idx_a * num_b + idx_b;
            *out = IoU3DWithCenterAndSize(box_a, box_b);

            idx += block_size;
        }
    }
}

void IoU3dCUDAKernel(const float *boxes_a,
                     const float *boxes_b,
                     float *iou,
                     int num_a,
                     int num_b) {
    int n = num_a * num_b;
    if (n == 0) {
        return;
    }
    int items_per_block = block_size * thread_size;
    int grid_size = (n + items_per_block - 1) / items_per_block;
    IoU3dElementKernel<<<grid_size, block_size>>>(boxes_a, boxes_b, iou, num_a,
                                                  num_b);
}

}  // namespace contrib
}  // namespace ml
}  // namespace open3d
