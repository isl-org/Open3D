#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//***************************************************************************************/
//
//    Based on PointRCNN Library (MIT License):
//    https://github.com/sshaoshuai/PointRCNN
//
//    Copyright (c) 2019 Shaoshuai Shi
//
//    Permission is hereby granted, free of charge, to any person obtaining a
//    copy of this software and associated documentation files (the "Software"),
//    to deal in the Software without restriction, including without limitation
//    the rights to use, copy, modify, merge, publish, distribute, sublicense,
//    and/or sell copies of the Software, and to permit persons to whom the
//    Software is furnished to do so, subject to the following conditions:
//
//    The above copyright notice and this permission notice shall be included in
//    all copies or substantial portions of the Software.
//
//    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
//    THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
//    FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
//    DEALINGS IN THE SOFTWARE.
//
//***************************************************************************************/

#include <math.h>
#include <stdio.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/ml/contrib/RoiPoolKernel.h"

namespace open3d {
namespace ml {
namespace contrib {

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__device__ inline int pt_in_box3d(float x,
                                  float y,
                                  float z,
                                  float cx,
                                  float bottom_y,
                                  float cz,
                                  float h,
                                  float w,
                                  float l,
                                  float angle,
                                  float max_dis) {
    float x_rot, z_rot, cosa, sina, cy;
    int in_flag;
    cy = bottom_y - h / 2.0;
    if ((fabsf(x - cx) > max_dis) || (fabsf(y - cy) > h / 2.0) ||
        (fabsf(z - cz) > max_dis)) {
        return 0;
    }
    cosa = cos(angle);
    sina = sin(angle);
    x_rot = (x - cx) * cosa + (z - cz) * (-sina);
    z_rot = (x - cx) * sina + (z - cz) * cosa;

    in_flag = (x_rot >= -l / 2.0) & (x_rot <= l / 2.0) & (z_rot >= -w / 2.0) &
              (z_rot <= w / 2.0);
    return in_flag;
}

__global__ void roipool3d_forward(int batch_size,
                                  int pts_num,
                                  int boxes_num,
                                  int feature_in_len,
                                  int sampled_pts_num,
                                  const float *xyz,
                                  const float *boxes3d,
                                  const float *pts_feature,
                                  float *pooled_features,
                                  int *pooled_empty_flag) {
    // params xyz: (B, N, 3)
    // params boxes3d: (B, M, 7)
    // params pts_feature: (B, N, C)
    // params pooled_features: (B, M, 512, 3+C)
    // params pooled_empty_flag: (B, M)

    int boxes_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boxes_idx >= boxes_num) {
        return;
    }

    for (int i = 0; i < batch_size; i++) {
        int cnt = 0;
        for (int k = 0; k < pts_num; k++) {
            int pt_offset = i * pts_num * 3 + k * 3;
            int box_offset = i * boxes_num * 7 + boxes_idx * 7;

            int cur_in_flag = pt_in_box3d(
                    xyz[pt_offset], xyz[pt_offset + 1], xyz[pt_offset + 2],
                    boxes3d[box_offset], boxes3d[box_offset + 1],
                    boxes3d[box_offset + 2], boxes3d[box_offset + 3],
                    boxes3d[box_offset + 4], boxes3d[box_offset + 5],
                    boxes3d[box_offset + 6], 10.0);
            if (cur_in_flag) {
                if (cnt < sampled_pts_num) {
                    int feature_out_offset =
                            i * boxes_num * sampled_pts_num *
                                    (3 + feature_in_len) +
                            boxes_idx * sampled_pts_num * (3 + feature_in_len) +
                            cnt * (3 + feature_in_len);

                    int feature_in_offset =
                            i * pts_num * feature_in_len + k * feature_in_len;

                    // copy xyz
                    for (int j = 0; j < 3; j++)
                        pooled_features[feature_out_offset + j] =
                                xyz[pt_offset + j];

                    // copy feature
                    for (int j = 0; j < feature_in_len; j++)
                        pooled_features[feature_out_offset + 3 + j] =
                                pts_feature[feature_in_offset + j];

                    cnt++;
                } else
                    break;
            }
        }

        if (cnt == 0) {
            pooled_empty_flag[i * boxes_num + boxes_idx] = 1;
        } else if (cnt < sampled_pts_num) {
            // duplicate same points for sampling
            for (int k = cnt; k < sampled_pts_num; k++) {
                int duplicate_idx = k % cnt;
                int src_offset =
                        i * boxes_num * sampled_pts_num * (3 + feature_in_len) +
                        boxes_idx * sampled_pts_num * (3 + feature_in_len) +
                        duplicate_idx * (3 + feature_in_len);
                int dst_offset =
                        i * boxes_num * sampled_pts_num * (3 + feature_in_len) +
                        boxes_idx * sampled_pts_num * (3 + feature_in_len) +
                        k * (3 + feature_in_len);
                for (int j = 0; j < 3 + feature_in_len; j++)
                    pooled_features[dst_offset + j] =
                            pooled_features[src_offset + j];
            }
        }
    }
}

__global__ void assign_pts_to_box3d(int batch_size,
                                    int pts_num,
                                    int boxes_num,
                                    const float *xyz,
                                    const float *boxes3d,
                                    int *pts_assign) {
    // params xyz: (B, N, 3)
    // params boxes3d: (B, M, 7)
    // params pts_assign: (B, N, M): idx of the corresponding box3d, -1 means
    // background points
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;

    if (pt_idx >= pts_num || box_idx >= boxes_num || bs_idx >= batch_size) {
        return;
    }
    int assign_idx =
            bs_idx * pts_num * boxes_num + pt_idx * boxes_num + box_idx;
    pts_assign[assign_idx] = 0;

    int box_offset = bs_idx * boxes_num * 7 + box_idx * 7;
    int pt_offset = bs_idx * pts_num * 3 + pt_idx * 3;

    int cur_in_flag =
            pt_in_box3d(xyz[pt_offset], xyz[pt_offset + 1], xyz[pt_offset + 2],
                        boxes3d[box_offset], boxes3d[box_offset + 1],
                        boxes3d[box_offset + 2], boxes3d[box_offset + 3],
                        boxes3d[box_offset + 4], boxes3d[box_offset + 5],
                        boxes3d[box_offset + 6], 10.0);

    pts_assign[assign_idx] = cur_in_flag;
    // printf("bs=%d, pt=%d, in=%d\n", bs_idx, pt_idx, pts_assign[bs_idx *
    // pts_num + pt_idx]);
}

__global__ void get_pooled_idx(int batch_size,
                               int pts_num,
                               int boxes_num,
                               int sampled_pts_num,
                               const int *pts_assign,
                               int *pts_idx,
                               int *pooled_empty_flag) {
    // params xyz: (B, N, 3)
    // params pts_feature: (B, N, C)
    // params pts_assign: (B, N)
    // params pts_idx: (B, M, 512)
    // params pooled_empty_flag: (B, M)

    int boxes_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boxes_idx >= boxes_num) {
        return;
    }

    int bs_idx = blockIdx.y;

    int cnt = 0;
    for (int k = 0; k < pts_num; k++) {
        if (pts_assign[bs_idx * pts_num * boxes_num + k * boxes_num +
                       boxes_idx]) {
            if (cnt < sampled_pts_num) {
                pts_idx[bs_idx * boxes_num * sampled_pts_num +
                        boxes_idx * sampled_pts_num + cnt] = k;
                cnt++;
            } else
                break;
        }
    }

    if (cnt == 0) {
        pooled_empty_flag[bs_idx * boxes_num + boxes_idx] = 1;
    } else if (cnt < sampled_pts_num) {
        // duplicate same points for sampling
        for (int k = cnt; k < sampled_pts_num; k++) {
            int duplicate_idx = k % cnt;
            int base_offset = bs_idx * boxes_num * sampled_pts_num +
                              boxes_idx * sampled_pts_num;
            pts_idx[base_offset + k] = pts_idx[base_offset + duplicate_idx];
        }
    }
}

__global__ void roipool3d_forward(int batch_size,
                                  int pts_num,
                                  int boxes_num,
                                  int feature_in_len,
                                  int sampled_pts_num,
                                  const float *xyz,
                                  const int *pts_idx,
                                  const float *pts_feature,
                                  float *pooled_features,
                                  int *pooled_empty_flag) {
    // params xyz: (B, N, 3)
    // params pts_idx: (B, M, 512)
    // params pts_feature: (B, N, C)
    // params pooled_features: (B, M, 512, 3+C)
    // params pooled_empty_flag: (B, M)

    int sample_pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;

    if (sample_pt_idx >= sampled_pts_num || box_idx >= boxes_num ||
        bs_idx >= batch_size) {
        return;
    }

    if (pooled_empty_flag[bs_idx * boxes_num + box_idx]) {
        return;
    }

    int temp_idx = bs_idx * boxes_num * sampled_pts_num +
                   box_idx * sampled_pts_num + sample_pt_idx;
    int src_pt_idx = pts_idx[temp_idx];
    int dst_feature_offset = temp_idx * (3 + feature_in_len);

    for (int j = 0; j < 3; j++)
        pooled_features[dst_feature_offset + j] =
                xyz[bs_idx * pts_num * 3 + src_pt_idx * 3 + j];

    int src_feature_offset =
            bs_idx * pts_num * feature_in_len + src_pt_idx * feature_in_len;
    for (int j = 0; j < feature_in_len; j++)
        pooled_features[dst_feature_offset + 3 + j] =
                pts_feature[src_feature_offset + j];
}

void roipool3dLauncher(int batch_size,
                       int pts_num,
                       int boxes_num,
                       int feature_in_len,
                       int sampled_pts_num,
                       const float *xyz,
                       const float *boxes3d,
                       const float *pts_feature,
                       float *pooled_features,
                       int *pooled_empty_flag) {
    // printf("batch_size=%d, pts_num=%d, boxes_num=%d\n", batch_size, pts_num,
    // boxes_num);
    int *pts_assign = NULL;
    hipMalloc(&pts_assign, batch_size * pts_num * boxes_num *
                                    sizeof(int));  // (batch_size, N, M)
    // hipMemset(&pts_assign, -1, batch_size * pts_num * boxes_num *
    // sizeof(int));

    dim3 blocks(DIVUP(pts_num, THREADS_PER_BLOCK), boxes_num,
                batch_size);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    assign_pts_to_box3d<<<blocks, threads>>>(batch_size, pts_num, boxes_num,
                                             xyz, boxes3d, pts_assign);

    int *pts_idx = NULL;
    hipMalloc(&pts_idx,
               batch_size * boxes_num * sampled_pts_num *
                       sizeof(int));  // (batch_size, M, sampled_pts_num)

    dim3 blocks2(DIVUP(boxes_num, THREADS_PER_BLOCK),
                 batch_size);  // blockIdx.x(col), blockIdx.y(row)
    get_pooled_idx<<<blocks2, threads>>>(batch_size, pts_num, boxes_num,
                                         sampled_pts_num, pts_assign, pts_idx,
                                         pooled_empty_flag);

    dim3 blocks_pool(DIVUP(sampled_pts_num, THREADS_PER_BLOCK), boxes_num,
                     batch_size);
    roipool3d_forward<<<blocks_pool, threads>>>(
            batch_size, pts_num, boxes_num, feature_in_len, sampled_pts_num,
            xyz, pts_idx, pts_feature, pooled_features, pooled_empty_flag);

    hipFree(pts_assign);
    hipFree(pts_idx);

#ifdef DEBUG
    core::cuda::Synchronize();  // for using printf in kernel function
#endif
}

}  // namespace contrib
}  // namespace ml
}  // namespace open3d
