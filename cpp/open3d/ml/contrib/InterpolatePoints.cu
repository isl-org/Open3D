#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "open3d/ml/contrib/InterpolatePoints.cuh"

namespace open3d {
namespace ml {
namespace contrib {

__global__ void three_nn_kernel(int b,
                                int n,
                                int m,
                                const float *__restrict__ unknown,
                                const float *__restrict__ known,
                                float *__restrict__ dist2,
                                int *__restrict__ idx) {
    // unknown: (B, N, 3)
    // known: (B, M, 3)
    // output:
    //      dist2: (B, N, 3)
    //      idx: (B, N, 3)

    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= n) return;

    unknown += bs_idx * n * 3 + pt_idx * 3;
    known += bs_idx * m * 3;
    dist2 += bs_idx * n * 3 + pt_idx * 3;
    idx += bs_idx * n * 3 + pt_idx * 3;

    float ux = unknown[0];
    float uy = unknown[1];
    float uz = unknown[2];

    double best1 = 1e40, best2 = 1e40, best3 = 1e40;
    int besti1 = 0, besti2 = 0, besti3 = 0;
    for (int k = 0; k < m; ++k) {
        float x = known[k * 3 + 0];
        float y = known[k * 3 + 1];
        float z = known[k * 3 + 2];
        float d =
                (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
        if (d < best1) {
            best3 = best2;
            besti3 = besti2;
            best2 = best1;
            besti2 = besti1;
            best1 = d;
            besti1 = k;
        } else if (d < best2) {
            best3 = best2;
            besti3 = besti2;
            best2 = d;
            besti2 = k;
        } else if (d < best3) {
            best3 = d;
            besti3 = k;
        }
    }
    dist2[0] = best1;
    dist2[1] = best2;
    dist2[2] = best3;
    idx[0] = besti1;
    idx[1] = besti2;
    idx[2] = besti3;
}

__global__ void three_interpolate_kernel(int b,
                                         int c,
                                         int m,
                                         int n,
                                         const float *__restrict__ points,
                                         const int *__restrict__ idx,
                                         const float *__restrict__ weight,
                                         float *__restrict__ out) {
    // points: (B, C, M)
    // idx: (B, N, 3)
    // weight: (B, N, 3)
    // output:
    //      out: (B, C, N)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;

    weight += bs_idx * n * 3 + pt_idx * 3;
    points += bs_idx * c * m + c_idx * m;
    idx += bs_idx * n * 3 + pt_idx * 3;
    out += bs_idx * c * n + c_idx * n;

    out[pt_idx] = weight[0] * points[idx[0]] + weight[1] * points[idx[1]] +
                  weight[2] * points[idx[2]];
}

__global__ void three_interpolate_grad_kernel(
        int b,
        int c,
        int n,
        int m,
        const float *__restrict__ grad_out,
        const int *__restrict__ idx,
        const float *__restrict__ weight,
        float *__restrict__ grad_points) {
    // grad_out: (B, C, N)
    // weight: (B, N, 3)
    // output:
    //      grad_points: (B, C, M)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;

    grad_out += bs_idx * c * n + c_idx * n + pt_idx;
    weight += bs_idx * n * 3 + pt_idx * 3;
    grad_points += bs_idx * c * m + c_idx * m;
    idx += bs_idx * n * 3 + pt_idx * 3;

    atomicAdd(grad_points + idx[0], grad_out[0] * weight[0]);
    atomicAdd(grad_points + idx[1], grad_out[0] * weight[1]);
    atomicAdd(grad_points + idx[2], grad_out[0] * weight[2]);
}

}  // namespace contrib
}  // namespace ml
}  // namespace open3d
