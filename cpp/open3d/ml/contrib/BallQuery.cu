#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "open3d/ml/contrib/BallQuery.cuh"

namespace open3d {
namespace ml {
namespace contrib {

__global__ void ball_query_kernel(int b,
                                  int n,
                                  int m,
                                  float radius,
                                  int nsample,
                                  const float *__restrict__ new_xyz,
                                  const float *__restrict__ xyz,
                                  int *__restrict__ idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                   (new_z - z) * (new_z - z);
        if (d2 < radius2) {
            if (cnt == 0) {
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample) break;
        }
    }
}

}  // namespace contrib
}  // namespace ml
}  // namespace open3d
