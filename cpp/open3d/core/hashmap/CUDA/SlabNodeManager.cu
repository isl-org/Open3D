#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

// Copyright 2019 Saman Ashkiani
// Rewritten by Wei Dong 2019 - 2020
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing permissions
// and limitations under the License.

#include "open3d/core/hashmap/CUDA/SlabNodeManager.h"

namespace open3d {
namespace core {

__global__ void CountSlabsPerSuperblockKernel(SlabNodeManagerImpl impl,
                                              uint32_t* slabs_per_superblock) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    int num_bitmaps = kBlocksPerSuperBlock * 32;
    if (tid >= num_bitmaps) {
        return;
    }

    for (uint32_t i = 0; i < kSuperBlocks; i++) {
        uint32_t read_bitmap = *(impl.get_ptr_for_bitmap(i, tid));
        atomicAdd(&slabs_per_superblock[i], __popc(read_bitmap));
    }
}
}  // namespace core
}  // namespace open3d
