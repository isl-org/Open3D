#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Dispatch.h"
#include "open3d/core/Indexer.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/BinaryEW.h"

namespace open3d {
namespace core {
namespace kernel {

// Cannot be a static function since on Windows a function enclosing
// __host__ __device__ lambda function must have external linkage.
template <typename src_t, typename dst_t, typename func_t>
void LaunchBinaryEWKernel(const Device& device,
                          const Indexer& indexer,
                          const func_t& element_kernel) {
    OPEN3D_ASSERT_HOST_DEVICE_LAMBDA(func_t);
    auto element_func = [=] OPEN3D_HOST_DEVICE(int64_t i) {
        element_kernel(indexer.GetInputPtr<src_t>(0, i),
                       indexer.GetInputPtr<src_t>(1, i),
                       indexer.GetOutputPtr<dst_t>(i));
    };
    ParallelFor(device, indexer.NumWorkloads(), element_func);
    OPEN3D_GET_LAST_CUDA_ERROR("LaunchBinaryEWKernel failed.");
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDAMaxElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = max(*static_cast<const scalar_t*>(lhs),
                                       *static_cast<const scalar_t*>(rhs));
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDAMinElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = min(*static_cast<const scalar_t*>(lhs),
                                       *static_cast<const scalar_t*>(rhs));
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDAAddElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = *static_cast<const scalar_t*>(lhs) +
                                   *static_cast<const scalar_t*>(rhs);
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDASubElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = *static_cast<const scalar_t*>(lhs) -
                                   *static_cast<const scalar_t*>(rhs);
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDAMulElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = *static_cast<const scalar_t*>(lhs) *
                                   *static_cast<const scalar_t*>(rhs);
}

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDADivElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<scalar_t*>(dst) = *static_cast<const scalar_t*>(lhs) /
                                   *static_cast<const scalar_t*>(rhs);
}

template <typename src_t, typename dst_t>
static OPEN3D_HOST_DEVICE void CUDALogicalAndElementKernel(const void* lhs,
                                                           const void* rhs,
                                                           void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            static_cast<bool>(*static_cast<const src_t*>(lhs)) &&
            static_cast<bool>(*static_cast<const src_t*>(rhs)));
}

template <typename src_t, typename dst_t>
static OPEN3D_HOST_DEVICE void CUDALogicalOrElementKernel(const void* lhs,
                                                          const void* rhs,
                                                          void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            static_cast<bool>(*static_cast<const src_t*>(lhs)) ||
            static_cast<bool>(*static_cast<const src_t*>(rhs)));
}

template <typename src_t, typename dst_t>
static OPEN3D_HOST_DEVICE void CUDALogicalXorElementKernel(const void* lhs,
                                                           const void* rhs,
                                                           void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            static_cast<bool>(*static_cast<const src_t*>(lhs)) !=
            static_cast<bool>(*static_cast<const src_t*>(rhs)));
}

template <typename src_t, typename dst_t>
static OPEN3D_HOST_DEVICE void CUDAGtElementKernel(const void* lhs,
                                                   const void* rhs,
                                                   void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) > *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
static OPEN3D_HOST_DEVICE void CUDALtElementKernel(const void* lhs,
                                                   const void* rhs,
                                                   void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) < *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
static void OPEN3D_HOST_DEVICE CUDAGeqElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) >= *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
static void OPEN3D_HOST_DEVICE CUDALeqElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) <= *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
static void OPEN3D_HOST_DEVICE CUDAEqElementKernel(const void* lhs,
                                                   const void* rhs,
                                                   void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) == *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
static void OPEN3D_HOST_DEVICE CUDANeqElementKernel(const void* lhs,
                                                    const void* rhs,
                                                    void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) != *static_cast<const src_t*>(rhs));
}

template <typename src_t, typename dst_t>
void LaunchBoolBinaryEWCUDAKernel(const Tensor& lhs,
                                  const Tensor& rhs,
                                  Tensor& dst,
                                  BinaryEWOpCode op_code,
                                  const Indexer& indexer) {
    Device device = lhs.GetDevice();
    switch (op_code) {
        case BinaryEWOpCode::LogicalAnd:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDALogicalAndElementKernel<src_t, dst_t>(lhs, rhs,
                                                                  dst);
                    });
            break;
        case BinaryEWOpCode::LogicalOr:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDALogicalOrElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::LogicalXor:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDALogicalXorElementKernel<src_t, dst_t>(lhs, rhs,
                                                                  dst);
                    });
            break;
        case BinaryEWOpCode::Gt:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDAGtElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::Lt:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDALtElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::Ge:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDAGeqElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::Le:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDALeqElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::Eq:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDAEqElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        case BinaryEWOpCode::Ne:
            LaunchBinaryEWKernel<src_t, dst_t>(
                    device, indexer,
                    [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                          void* dst) {
                        CUDANeqElementKernel<src_t, dst_t>(lhs, rhs, dst);
                    });
            break;
        default:
            break;
    }
}

void BinaryEWCUDA(const Tensor& lhs,
                  const Tensor& rhs,
                  Tensor& dst,
                  BinaryEWOpCode op_code) {
    // It has been checked that
    // - lhs, rhs, dst are all in the same CUDA device
    // - lhs, rhs have the same dtype, dst also has the same dtype or is boolean
    Device src_device = lhs.GetDevice();
    Dtype src_dtype = lhs.GetDtype();
    Dtype dst_dtype = dst.GetDtype();

    CUDAScopedDevice scoped_device(src_device);

    if (s_boolean_binary_ew_op_codes.find(op_code) !=
        s_boolean_binary_ew_op_codes.end()) {
        DISPATCH_DTYPE_TO_TEMPLATE_WITH_BOOL(src_dtype, [&]() {
            if (dst_dtype == src_dtype) {
                // Inplace boolean op's output type is the same as the
                // input. e.g. np.logical_and(a, b, out=a), where a, b are
                // floats.
                Indexer indexer({lhs, rhs}, dst, DtypePolicy::ALL_SAME);
                LaunchBoolBinaryEWCUDAKernel<scalar_t, scalar_t>(
                        lhs, rhs, dst, op_code, indexer);
            } else if (dst_dtype == core::Bool) {
                // By default, output is boolean type.
                Indexer indexer({lhs, rhs}, dst,
                                DtypePolicy::INPUT_SAME_OUTPUT_BOOL);

                LaunchBoolBinaryEWCUDAKernel<scalar_t, bool>(lhs, rhs, dst,
                                                             op_code, indexer);
            } else {
                utility::LogError(
                        "Boolean op's output type must be boolean or the "
                        "same type as the input.");
            }
        });
    } else if (op_code == BinaryEWOpCode::Maximum ||
               op_code == BinaryEWOpCode::Minimum) {
        Indexer indexer({lhs, rhs}, dst, DtypePolicy::ALL_SAME);
        DISPATCH_DTYPE_TO_TEMPLATE_WITH_BOOL(src_dtype, [&]() {
            switch (op_code) {
                case BinaryEWOpCode::Maximum:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDAMaxElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                case BinaryEWOpCode::Minimum:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDAMinElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                default:
                    break;
            }
        });
    } else {
        Indexer indexer({lhs, rhs}, dst, DtypePolicy::ALL_SAME);
        DISPATCH_DTYPE_TO_TEMPLATE(src_dtype, [&]() {
            switch (op_code) {
                case BinaryEWOpCode::Add:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDAAddElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                case BinaryEWOpCode::Sub:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDASubElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                case BinaryEWOpCode::Mul:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDAMulElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                case BinaryEWOpCode::Div:
                    LaunchBinaryEWKernel<scalar_t, scalar_t>(
                            src_device, indexer,
                            [] OPEN3D_HOST_DEVICE(const void* lhs, void* rhs,
                                                  void* dst) {
                                CUDADivElementKernel<scalar_t>(lhs, rhs, dst);
                            });
                    break;
                default:
                    break;
            }
        });
    }
}

}  // namespace kernel
}  // namespace core
}  // namespace open3d
