// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------
//

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/nns/FixedRadiusIndex.h"
#include "open3d/core/nns/FixedRadiusSearchImpl.cuh"
#include "open3d/core/nns/NeighborSearchAllocator.h"
#include "open3d/core/nns/NeighborSearchCommon.h"

namespace open3d {
namespace core {
namespace nns {

template <class T>
void BuildSpatialHashTableCUDA(const Tensor& points,
                               double radius,
                               const Tensor& points_row_splits,
                               const Tensor& hash_table_splits,
                               Tensor& hash_table_index,
                               Tensor& hash_table_cell_splits) {
    CUDAScopedDevice scoped_device(points.GetDevice());
    const hipStream_t stream = 0;
    int texture_alignment = 512;

    void* temp_ptr = nullptr;
    size_t temp_size = 0;

    impl::BuildSpatialHashTableCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            points.GetShape()[0], points.GetDataPtr<T>(), T(radius),
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>());

    Device device = points.GetDevice();
    Tensor temp_tensor =
            Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
    temp_ptr = temp_tensor.GetDataPtr();

    impl::BuildSpatialHashTableCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            points.GetShape()[0], points.GetDataPtr<T>(), T(radius),
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>());
}

template <class T, class TIndex>
void FixedRadiusSearchCUDA(const Tensor& points,
                           const Tensor& queries,
                           double radius,
                           const Tensor& points_row_splits,
                           const Tensor& queries_row_splits,
                           const Tensor& hash_table_splits,
                           const Tensor& hash_table_index,
                           const Tensor& hash_table_cell_splits,
                           const Metric metric,
                           const bool ignore_query_point,
                           const bool return_distances,
                           const bool sort,
                           Tensor& neighbors_index,
                           Tensor& neighbors_row_splits,
                           Tensor& neighbors_distance) {
    CUDAScopedDevice scoped_device(points.GetDevice());
    const hipStream_t stream = 0;
    int texture_alignment = 512;

    Device device = points.GetDevice();
    Dtype dtype = points.GetDtype();
    Dtype index_dtype = Dtype::FromType<TIndex>();

    NeighborSearchAllocator<T, TIndex> output_allocator(device);
    void* temp_ptr = nullptr;
    size_t temp_size = 0;

    impl::FixedRadiusSearchCUDA<T, TIndex>(
            stream, temp_ptr, temp_size, texture_alignment,
            neighbors_row_splits.GetDataPtr<int64_t>(), points.GetShape()[0],
            points.GetDataPtr<T>(), queries.GetShape()[0],
            queries.GetDataPtr<T>(), T(radius), points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, ignore_query_point,
            return_distances, output_allocator);

    Tensor temp_tensor =
            Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
    temp_ptr = temp_tensor.GetDataPtr();

    impl::FixedRadiusSearchCUDA<T, TIndex>(
            stream, temp_ptr, temp_size, texture_alignment,
            neighbors_row_splits.GetDataPtr<int64_t>(), points.GetShape()[0],
            points.GetDataPtr<T>(), queries.GetShape()[0],
            queries.GetDataPtr<T>(), T(radius), points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, ignore_query_point,
            return_distances, output_allocator);

    Tensor indices_unsorted = output_allocator.NeighborsIndex();
    Tensor distances_unsorted = output_allocator.NeighborsDistance();

    if (!sort) {
        neighbors_index = indices_unsorted;
        neighbors_distance = distances_unsorted;
    } else {
        // Sort indices & distances.
        temp_ptr = nullptr;
        temp_size = 0;

        int64_t num_indices = indices_unsorted.GetShape()[0];
        int64_t num_segments = neighbors_row_splits.GetShape()[0] - 1;
        Tensor indices_sorted =
                Tensor::Empty({num_indices}, index_dtype, device);
        Tensor distances_sorted = Tensor::Empty({num_indices}, dtype, device);

        // Determine temp_size for sorting
        impl::SortPairs(temp_ptr, temp_size, texture_alignment, num_indices,
                        num_segments,
                        neighbors_row_splits.GetDataPtr<int64_t>(),
                        indices_unsorted.GetDataPtr<TIndex>(),
                        distances_unsorted.GetDataPtr<T>(),
                        indices_sorted.GetDataPtr<TIndex>(),
                        distances_sorted.GetDataPtr<T>());

        temp_tensor = Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
        temp_ptr = temp_tensor.GetDataPtr();

        // Actually run the sorting.
        impl::SortPairs(temp_ptr, temp_size, texture_alignment, num_indices,
                        num_segments,
                        neighbors_row_splits.GetDataPtr<int64_t>(),
                        indices_unsorted.GetDataPtr<TIndex>(),
                        distances_unsorted.GetDataPtr<T>(),
                        indices_sorted.GetDataPtr<TIndex>(),
                        distances_sorted.GetDataPtr<T>());
        neighbors_index = indices_sorted;
        neighbors_distance = distances_sorted;
    }
}

template <class T, class TIndex>
void HybridSearchCUDA(const Tensor& points,
                      const Tensor& queries,
                      double radius,
                      int max_knn,
                      const Tensor& points_row_splits,
                      const Tensor& queries_row_splits,
                      const Tensor& hash_table_splits,
                      const Tensor& hash_table_index,
                      const Tensor& hash_table_cell_splits,
                      const Metric metric,
                      Tensor& neighbors_index,
                      Tensor& neighbors_count,
                      Tensor& neighbors_distance) {
    CUDAScopedDevice scoped_device(points.GetDevice());
    const hipStream_t stream = 0;

    Device device = points.GetDevice();

    NeighborSearchAllocator<T, TIndex> output_allocator(device);

    impl::HybridSearchCUDA<T, TIndex>(
            stream, points.GetShape()[0], points.GetDataPtr<T>(),
            queries.GetShape()[0], queries.GetDataPtr<T>(), T(radius), max_knn,
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, output_allocator);

    neighbors_index = output_allocator.NeighborsIndex();
    neighbors_distance = output_allocator.NeighborsDistance();
    neighbors_count = output_allocator.NeighborsCount();
}

#define INSTANTIATE_BUILD(T)                                                  \
    template void BuildSpatialHashTableCUDA<T>(                               \
            const Tensor& points, double radius,                              \
            const Tensor& points_row_splits, const Tensor& hash_table_splits, \
            Tensor& hash_table_index, Tensor& hash_table_cell_splits);

#define INSTANTIATE_RADIUS(T, TIndex)                                          \
    template void FixedRadiusSearchCUDA<T, TIndex>(                            \
            const Tensor& points, const Tensor& queries, double radius,        \
            const Tensor& points_row_splits, const Tensor& queries_row_splits, \
            const Tensor& hash_table_splits, const Tensor& hash_table_index,   \
            const Tensor& hash_table_cell_splits, const Metric metric,         \
            const bool ignore_query_point, const bool return_distances,        \
            const bool sort, Tensor& neighbors_index,                          \
            Tensor& neighbors_row_splits, Tensor& neighbors_distance);

#define INSTANTIATE_HYBRID(T, TIndex)                                          \
    template void HybridSearchCUDA<T, TIndex>(                                 \
            const Tensor& points, const Tensor& queries, double radius,        \
            int max_knn, const Tensor& points_row_splits,                      \
            const Tensor& queries_row_splits, const Tensor& hash_table_splits, \
            const Tensor& hash_table_index,                                    \
            const Tensor& hash_table_cell_splits, const Metric metric,         \
            Tensor& neighbors_index, Tensor& neighbors_count,                  \
            Tensor& neighbors_distance);

INSTANTIATE_BUILD(float)
INSTANTIATE_BUILD(double)

INSTANTIATE_RADIUS(float, int32_t)
INSTANTIATE_RADIUS(float, int64_t)
INSTANTIATE_RADIUS(double, int32_t)
INSTANTIATE_RADIUS(double, int64_t)

INSTANTIATE_HYBRID(float, int32_t)
INSTANTIATE_HYBRID(float, int64_t)
INSTANTIATE_HYBRID(double, int32_t)
INSTANTIATE_HYBRID(double, int64_t)

}  // namespace nns
}  // namespace core
}  // namespace open3d
