#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.open3d.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "open3d/t/geometry/kernel/PointCloudImpl.h"

namespace open3d {
namespace t {
namespace geometry {
namespace kernel {
namespace pointcloud {

void ProjectCUDA(
        core::Tensor& depth,
        utility::optional<std::reference_wrapper<core::Tensor>> image_colors,
        const core::Tensor& points,
        utility::optional<std::reference_wrapper<const core::Tensor>> colors,
        const core::Tensor& intrinsics,
        const core::Tensor& extrinsics,
        float depth_scale,
        float depth_max) {
    const bool has_colors = image_colors.has_value();

    int64_t n = points.GetLength();

    const float* points_ptr = points.GetDataPtr<float>();
    const float* point_colors_ptr =
            has_colors ? colors.value().get().GetDataPtr<float>() : nullptr;

    TransformIndexer transform_indexer(intrinsics, extrinsics, 1.0f);
    NDArrayIndexer depth_indexer(depth, 2);

    // Pass 1: depth map
    core::ParallelFor(
            depth.GetDevice(), n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float x = points_ptr[3 * workload_idx + 0];
                float y = points_ptr[3 * workload_idx + 1];
                float z = points_ptr[3 * workload_idx + 2];

                // coordinate in camera (in voxel -> in meter)
                float xc, yc, zc, u, v;
                transform_indexer.RigidTransform(x, y, z, &xc, &yc, &zc);

                // coordinate in image (in pixel)
                transform_indexer.Project(xc, yc, zc, &u, &v);
                u = round(u);
                v = round(v);
                if (!depth_indexer.InBoundary(u, v) || zc <= 0 ||
                    zc > depth_max) {
                    return;
                }

                float* depth_ptr = depth_indexer.GetDataPtr<float>(
                        static_cast<int64_t>(u), static_cast<int64_t>(v));
                float d = zc * depth_scale;
                float d_old = atomicExch(depth_ptr, d);
                if (d_old > 0) {
                    atomicMinf(depth_ptr, d_old);
                }
            });

    // Pass 2: color map
    if (!has_colors) return;

    NDArrayIndexer color_indexer(image_colors.value().get(), 2);
    float precision_bound = depth_scale * 1e-4;
    core::ParallelFor(
            depth.GetDevice(), n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float x = points_ptr[3 * workload_idx + 0];
                float y = points_ptr[3 * workload_idx + 1];
                float z = points_ptr[3 * workload_idx + 2];

                // coordinate in camera (in voxel -> in meter)
                float xc, yc, zc, u, v;
                transform_indexer.RigidTransform(x, y, z, &xc, &yc, &zc);

                // coordinate in image (in pixel)
                transform_indexer.Project(xc, yc, zc, &u, &v);
                if (!depth_indexer.InBoundary(u, v) || zc <= 0 ||
                    zc > depth_max) {
                    return;
                }

                float dmap = *depth_indexer.GetDataPtr<float>(
                        static_cast<int64_t>(u), static_cast<int64_t>(v));
                float d = zc * depth_scale;
                if (d < dmap + precision_bound) {
                    float* color_ptr = color_indexer.GetDataPtr<float>(
                            static_cast<int64_t>(u), static_cast<int64_t>(v));
                    color_ptr[0] = point_colors_ptr[3 * workload_idx + 0];
                    color_ptr[1] = point_colors_ptr[3 * workload_idx + 1];
                    color_ptr[2] = point_colors_ptr[3 * workload_idx + 2];
                }
            });
}

}  // namespace pointcloud
}  // namespace kernel
}  // namespace geometry
}  // namespace t
}  // namespace open3d
